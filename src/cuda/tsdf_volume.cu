#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"
#include "utils.hpp"

using namespace pcl::device;

namespace pcl
{
  namespace device
  {
    template<typename T>
    __global__ void
    initializeVolume (PtrStepSz<T> volume)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x < volume.cols && y < volume.rows)
        pack_tsdf (0.f, 0, (short2&)volume.ptr (y)[x]);
    }

    template<typename T>
    void
    initVolume (PtrStepSz<T> volume)
    {
      dim3 block (32, 16);
      dim3 grid (1, 1, 1);
      grid.x = divUp (volume.cols, block.x);
      grid.y = divUp (volume.rows, block.y);

      initializeVolume << < grid, block >> > (volume);
      cudaSafeCall ( hipGetLastError () );
      cudaSafeCall (hipDeviceSynchronize ());
    }
  }
}


namespace pcl
{
  namespace device
  {
    struct Tsdf
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8,
        MAX_WEIGHT = 1 << 7
      };
	};
  }
}

template void pcl::device::initVolume (PtrStepSz<short2> volume);
template void pcl::device::initVolume (PtrStepSz<ushort2> volume);

namespace pcl
{
  namespace device
  {
    __global__ void
    scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= depth.cols || y >= depth.rows)
        return;

      int Dp = depth.ptr (y)[x];

      float xl = (x - intr.cx) / intr.fx;
      float yl = (y - intr.cy) / intr.fy;
      float lambda = sqrtf (xl * xl + yl * yl + 1);

      scaled.ptr (y)[x] = Dp * lambda;
    }

    __global__ void
    tsdf23 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
            const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step)
      {
        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x];

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if (Dp_scaled != 0 && sdf >= -tranc_dist)
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);

            const int Wrk = 1;
						
			float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            pack_tsdf (tsdf_new, weight_new, *pos);
          }
        }
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__

  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//We are using it
void
pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                  const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                  float tranc_dist,
                                  PtrStep<short2> volume, DeviceArray2D<float>& depthScaled)
{
  depthScaled.create (depth.rows, depth.cols);

  dim3 block_scale (32, 8);
  dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

  scaleDepth << < grid_scale, block_scale >> > (depth, depthScaled, intr);
  cudaSafeCall ( hipGetLastError () );

  float3 cell_size;
  cell_size.x = volume_size.x / VOLUME_X;
  cell_size.y = volume_size.y / VOLUME_Y;
  cell_size.z = volume_size.z / VOLUME_Z;

  //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 block (16, 16);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

  tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);    
  //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);

  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

namespace pcl
{
  namespace device
  {
    __global__ void
    tsdf24 (const PtrStepSz<float> depthScaled, PtrStep<ushort2> volume,
            const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
      const int x = threadIdx.x + blockIdx.x * blockDim.x;
      const int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      const float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      const float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      const float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      const float tranc_dist_inv = 1.0f / tranc_dist;

      ushort2* voxel = volume.ptr (y) + x;
      const int elem_step = volume.step * VOLUME_Y / sizeof(ushort2);
      const ushort2* end_voxel = volume.ptr (y + VOLUME_Y * VOLUME_Z);

      while (voxel < end_voxel)
      {
        float inv_z = 1.0f / __fmaf_rn (Rcurr_inv.data[2].z, z_scaled, v_z);

        // project to current cam
        int2 coo =
        {
          __float2int_rn (__fmaf_rn (v_x, inv_z, intr.cx)),
          __float2int_rn (__fmaf_rn (v_y, inv_z, intr.cy))
        };

        if (inv_z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x];

          float sdf = Dp_scaled - sqrtf (__fmaf_rn (v_g_z, v_g_z, v_g_part_norm));

          if (Dp_scaled != 0 && sdf >= -tranc_dist)
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            float tsdf_prev;
            int weight_prev;
            unpack_tsdf ((short2&)*voxel, tsdf_prev, weight_prev);

            //const int Wrk = 1;

            float tsdf_new = __fmaf_rn (tsdf_prev, weight_prev, tsdf) / (weight_prev + 1);
            int weight_new = min (weight_prev + 1, Tsdf::MAX_WEIGHT);

            pack_tsdf (tsdf_new, weight_new, (short2&)*voxel);
          }
        }

        v_g_z += cell_size.z;

        z_scaled += cell_size.z;

        v_x += Rcurr_inv_0_z_scaled;
        v_y += Rcurr_inv_1_z_scaled;

        voxel += elem_step;
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__

  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                  const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                  float tranc_dist,
                                  PtrStep<ushort2> volume, DeviceArray2D<float>& depthRawScaled)
{
  depthRawScaled.create (depth.rows, depth.cols);
  {
    dim3 block (32, 8);
    dim3 grid (divUp (depth.cols, block.x), divUp (depth.rows, block.y));

    scaleDepth << < grid, block >> > (depth, depthRawScaled, intr);
    cudaSafeCall ( hipGetLastError () );
  }

  {
    float3 cell_size;
    cell_size.x = volume_size.x / VOLUME_X;
    cell_size.y = volume_size.y / VOLUME_Y;
    cell_size.z = volume_size.z / VOLUME_Z;

    dim3 block (Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
    dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(tsdf24), hipFuncCachePreferL1);

    tsdf24 << < grid, block >> > (depthRawScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);
    cudaSafeCall (hipGetLastError ());
  }

  cudaSafeCall (hipDeviceSynchronize ());
}
