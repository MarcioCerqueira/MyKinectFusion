#include "hip/hip_runtime.h"
#include "VolumeRendering/TriCubicInterpolationPreFilter.h"

inline __device__ __host__ uint UMIN(uint a, uint b)
{
	return a < b ? a : b;
}

inline __device__ __host__ uint PowTwoDivider(uint n)
{
	if (n == 0) return 0;
	uint divider = 1;
	while ((n & divider) == 0) divider <<= 1; 
	return divider;
}

#define Pole (sqrt(3.0f)-2.0f)  //pole for cubic b-spline

template<class floatN>
__host__ __device__ floatN InitialCausalCoefficient(
	floatN* c,			// coefficients
	uint DataLength,	// number of coefficients
	int step)			// element interleave in bytes
{
	const uint Horizon = UMIN(12, DataLength);

	// this initialization corresponds to clamping boundaries
	// accelerated loop
	float zn = Pole;
	floatN Sum = *c;
	for (uint n = 0; n < Horizon; n++) {
		Sum += zn * *c;
		zn *= Pole;
		c = (floatN*)((uchar*)c + step);
	}
	return(Sum);
}

template<class floatN>
__host__ __device__ floatN InitialAntiCausalCoefficient(
	floatN* c,			// last coefficient
	uint DataLength,	// number of samples or coefficients
	int step)			// element interleave in bytes
{
	// this initialization corresponds to clamping boundaries
	return((Pole / (Pole - 1.0f)) * *c);
}

template<class floatN>
__host__ __device__ void ConvertToInterpolationCoefficients(
	floatN* coeffs,		// input samples --> output coefficients
	uint DataLength,	// number of samples or coefficients
	int step)			// element interleave in bytes
{
	// compute the overall gain
	const float Lambda = (1.0f - Pole) * (1.0f - 1.0f / Pole);

	// causal initialization
	floatN* c = coeffs;
	floatN previous_c;  //cache the previously calculated c rather than look it up again (faster!)
	*c = previous_c = Lambda * InitialCausalCoefficient(c, DataLength, step);
	// causal recursion
	for (uint n = 1; n < DataLength; n++) {
		c = (floatN*)((uchar*)c + step);
		*c = previous_c = Lambda * *c + Pole * previous_c;
	}
	// anticausal initialization
	*c = previous_c = InitialAntiCausalCoefficient(c, DataLength, step);
	// anticausal recursion
	for (int n = DataLength - 2; 0 <= n; n--) {
		c = (floatN*)((uchar*)c - step);
		*c = previous_c = Pole * (previous_c - *c);
	}
}

template<class floatN>
__global__ void SamplesToCoefficients3DX(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in x-direction
	const uint y = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = (z * height + y) * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr, width, sizeof(floatN));
}

template<class floatN>
__global__ void SamplesToCoefficients3DY(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in y-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = z * height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr + x, height, pitch);
}

template<class floatN>
__global__ void SamplesToCoefficients3DZ(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in z-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = y * pitch;
	const uint slice = height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr + x, depth, slice);
}


//! Convert the voxel values into cubic b-spline coefficients
//! @param volume  pointer to the voxel volume in GPU (device) memory
//! @param pitch   width in bytes (including padding bytes)
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
template<class floatN>
extern void CubicBSplinePrefilter3D2(floatN* volume, uint pitch, uint width, uint height, uint depth)
{
	// Try to determine the optimal block dimensions
	uint dimX = min(min(PowTwoDivider(width), PowTwoDivider(height)), 64);
	uint dimY = min(min(PowTwoDivider(depth), PowTwoDivider(height)), 512/dimX);
	dim3 dimBlock(dimX, dimY);

	// Replace the voxel values by the b-spline coefficients
	dim3 dimGridX(height / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DX<floatN><<<dimGridX, dimBlock>>>(volume, pitch, width, height, depth);
	//CUT_CHECK_ERROR("SamplesToCoefficients3DX kernel failed");

	dim3 dimGridY(width / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DY<floatN><<<dimGridY, dimBlock>>>(volume, pitch, width, height, depth);
	//CUT_CHECK_ERROR("SamplesToCoefficients3DY kernel failed");

	dim3 dimGridZ(width / dimBlock.x, height / dimBlock.y);
	SamplesToCoefficients3DZ<floatN><<<dimGridZ, dimBlock>>>(volume, pitch, width, height, depth);
	//CUT_CHECK_ERROR("SamplesToCoefficients3DZ kernel failed");
}

void CubicBSplinePrefilter3D(float* volume, uint pitch, uint width, uint height, uint depth) {
	CubicBSplinePrefilter3D2(volume, pitch, width, height, depth);
}