#include "hip/hip_runtime.h"
#include "VolumeRendering/memcpy.h"

template<class T> __device__ float Multiplier()	{ return 1.0f; }
template<> __device__ float Multiplier<unsigned char>()	{ return 255.0f; }
template<> __device__ float Multiplier<signed char>()	{ return 127.0f; }
template<> __device__ float Multiplier<unsigned short>(){ return 65535.0f; }
template<> __device__ float Multiplier<short>()	{ return 32767.0f; }

inline __device__ __host__ uint PowTwoDivider2(uint n)
{
	if (n == 0) return 0;
	uint divider = 1;
	while ((n & divider) == 0) divider <<= 1; 
	return divider;
}

template<class T> __global__ void CopyCast(unsigned char* destination, T* source, unsigned int pitch, unsigned int width)
{
	uint2 index = make_uint2(
		__umul24(blockIdx.x, blockDim.x) + threadIdx.x,
		__umul24(blockIdx.y, blockDim.y) + threadIdx.y);

	float* dest = (float*)(destination + index.y * pitch) + index.x;
	*dest = (1.0f/Multiplier<T>()) * (float)(source[index.y * width + index.x]);
}

template<class T> __global__ void CopyCastBack(T* destination, uchar* source, uint pitch, uint width)
{
	uint2 index = make_uint2(
		__umul24(blockIdx.x, blockDim.x) + threadIdx.x,
		__umul24(blockIdx.y, blockDim.y) + threadIdx.y);

	float* src = (float*)(source + index.y * pitch) + index.x;
	destination[index.y * width + index.x] = (T)(Multiplier<T>() * *src);
}

//! Allocate GPU memory and copy a voxel volume from CPU to GPU memory
//! and cast it to the normalized floating point format
//! @return the pointer to the GPU copy of the voxel volume
//! @param host  pointer to the voxel volume in CPU (host) memory
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
template<class T> extern hipPitchedPtr CastVolumeHostToDevice2(T* host, unsigned int width, unsigned int height, unsigned int depth)
{
	hipPitchedPtr device = {0};
	const hipExtent extent = make_hipExtent(width * sizeof(float), height, depth);
	hipMalloc3D(&device, extent);
	const size_t pitchedBytesPerSlice = device.pitch * device.ysize;
	
	T* temp = 0;
	const unsigned int voxelsPerSlice = width * height;
	const size_t nrOfBytesTemp = voxelsPerSlice * sizeof(T);
	hipMalloc((void**)&temp, nrOfBytesTemp);

	unsigned int dimX = min(PowTwoDivider2(width), 64);
	dim3 dimBlock(dimX, min(PowTwoDivider2(height), 512 / dimX));
	dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
	size_t offsetHost = 0;
	size_t offsetDevice = 0;
	
	for (unsigned int slice = 0; slice < depth; slice++)
	{
		hipMemcpy(temp, host + offsetHost, nrOfBytesTemp, hipMemcpyHostToDevice);
		CopyCast<T><<<dimGrid, dimBlock>>>((unsigned char*)device.ptr + offsetDevice, temp, (unsigned int)device.pitch, width);
		//CUT_CHECK_ERROR("Cast kernel failed");
		offsetHost += voxelsPerSlice;
		offsetDevice += pitchedBytesPerSlice;
	}

	hipFree(temp);  //free the temp GPU volume
	return device;
}

//! Copy a voxel volume from GPU to CPU memory
//! while casting it to the desired format
//! @param host  pointer to the voxel volume in CPU (host) memory
//! @param device  pitched pointer to the voxel volume in GPU (device) memory
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
//! @note The \host CPU memory should be pre-allocated
template<class T> extern void CastVolumeDeviceToHost2(T* host, const hipPitchedPtr device, uint width, uint height, uint depth)
{
	T* temp = 0;
	const uint voxelsPerSlice = width * height;
	const size_t nrOfBytesTemp = voxelsPerSlice * sizeof(T);
	hipMalloc((void**)&temp, nrOfBytesTemp);

	uint dimX = min(PowTwoDivider2(width), 64);
	dim3 dimBlock(dimX, min(PowTwoDivider2(height), 512 / dimX));
	dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
	const size_t pitchedBytesPerSlice = device.pitch * device.ysize;
	size_t offsetHost = 0;
	size_t offsetDevice = 0;
	
	for (uint slice = 0; slice < depth; slice++)
	{
		CopyCastBack<T><<<dimGrid, dimBlock>>>(temp, (uchar*)device.ptr + offsetDevice, (uint)device.pitch, width);
		//CUT_CHECK_ERROR("Cast kernel failed");
		hipMemcpy(host + offsetHost, temp, nrOfBytesTemp, hipMemcpyDeviceToHost);
		offsetHost += voxelsPerSlice;
		offsetDevice += pitchedBytesPerSlice;
	}

	hipFree(temp);  //free the temp GPU volume
}

hipPitchedPtr CastVolumeHostToDevice(unsigned char* host, unsigned int width, unsigned int height, unsigned int depth) {
	return CastVolumeHostToDevice2(host, width, height, depth);
}

void CastVolumeDeviceToHost(unsigned char* host, const hipPitchedPtr device, uint width, uint height, uint depth) {
	CastVolumeDeviceToHost2(host, device, width, height, depth);
}