#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

using namespace pcl::device;
using namespace pcl::gpu;

namespace pcl
{
  namespace device
  {
    __global__ void
    computeVmapKernel (const PtrStepSz<unsigned short> depth, PtrStep<float> vmap, float fx_inv, float fy_inv, float cx, float cy)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u < depth.cols && v < depth.rows)
      {
        int z = depth.ptr (v)[u];

        if (z != 0)
        {
          float vx = z * (u - cx) * fx_inv;
          float vy = z * (v - cy) * fy_inv;
          float vz = z;

          vmap.ptr (v                 )[u] = vx;
          vmap.ptr (v + depth.rows    )[u] = vy;
          vmap.ptr (v + depth.rows * 2)[u] = vz;
        }
        else
          vmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();

      }
    }

    __global__ void
    computeNmapKernel (int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      if (u == cols - 1 || v == rows - 1)
      {
        nmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();
        return;
      }

      float3 v00, v01, v10;
      v00.x = vmap.ptr (v  )[u];
      v01.x = vmap.ptr (v  )[u + 1];
      v10.x = vmap.ptr (v + 1)[u];

      if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))
      {
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];
        v10.y = vmap.ptr (v + 1 + rows)[u];

        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
      }
      else
        nmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createVMap (const Intr& intr, const DepthMap& depth, MapArr& vmap)
{
  vmap.create (depth.rows () * 3, depth.cols ());

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (depth.cols (), block.x);
  grid.y = divUp (depth.rows (), block.y);

  float fx = intr.fx, cx = intr.cx;
  float fy = intr.fy, cy = intr.cy;

  computeVmapKernel << < grid, block >> > (depth, vmap, 1.f / fx, 1.f / fy, cx, cy);
  cudaSafeCall (hipGetLastError ());
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createNMap (const MapArr& vmap, MapArr& nmap)
{
  nmap.create (vmap.rows (), vmap.cols ());

  int rows = vmap.rows () / 3;
  int cols = vmap.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  computeNmapKernel << < grid, block >> > (rows, cols, vmap, nmap);
  cudaSafeCall (hipGetLastError ());
}

namespace pcl
{
  namespace device
  {
    __global__ void
    tranformMapsKernel (int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                        const Mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst, bool inverse)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      const float qnan = pcl::device::numeric_limits<float>::quiet_NaN ();

      if (x < cols && y < rows)
      {
        //vetexes
        float3 vsrc, vdst = make_float3 (qnan, qnan, qnan);
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
          vsrc.y = vmap_src.ptr (y + rows)[x];
          vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

		  if(!inverse)
			vdst = Rmat * vsrc + tvec;
		  else
			vdst = Rmat * (vsrc - tvec);

          vmap_dst.ptr (y + rows)[x] = vdst.y;
          vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (qnan, qnan, qnan);
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
          nsrc.y = nmap_src.ptr (y + rows)[x];
          nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

          ndst = Rmat * nsrc;

          nmap_dst.ptr (y + rows)[x] = ndst.y;
          nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
      }
    }

	__global__ void
    tranformMapsKernel (int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                        const Mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst, 
						const float3 newOrigin, const float3 objectCenter)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      const float qnan = pcl::device::numeric_limits<float>::quiet_NaN ();

      if (x < cols && y < rows)
      {
        //vetexes
        float3 vsrc, vdst = make_float3 (qnan, qnan, qnan);
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
			vsrc.y = vmap_src.ptr (y + rows)[x];
			vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

			if(vsrc.z != 0) 
			{
			  //Convert to object's center
			  vsrc.x -= newOrigin.x;
			  vsrc.y -= newOrigin.y;
			  vsrc.z -= newOrigin.z;

			  vsrc.x -= objectCenter.x;
			  vsrc.y -= objectCenter.y;
			  vsrc.z -= objectCenter.z;

			  vdst = Rmat * vsrc + tvec;

			  /*
			  if(!inverseTransform) 
				vdst = Rmat * vsrc + tvec;
		      else
				vdst = Rmat * (vsrc - tvec);
			  */
			  
			  
			  vdst.x += objectCenter.x;
			  vdst.y += objectCenter.y;
			  vdst.z += objectCenter.z;

			  vdst.x += newOrigin.x;
			  vdst.y += newOrigin.y;
			  vdst.z += newOrigin.z;

		    
		     }

		  vmap_dst.ptr (y + rows)[x] = vdst.y;
		  vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
		
		}

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (qnan, qnan, qnan);
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
          nsrc.y = nmap_src.ptr (y + rows)[x];
          nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

          ndst = Rmat * nsrc;

          nmap_dst.ptr (y + rows)[x] = ndst.y;
          nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
      }
    }

  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::tranformMaps (const MapArr& vmap_src, const MapArr& nmap_src, 
                           const Mat33& Rmat, const float3& tvec, 
                           MapArr& vmap_dst, MapArr& nmap_dst, bool inverse)
{
  int cols = vmap_src.cols ();
  int rows = vmap_src.rows () / 3;

  vmap_dst.create (rows * 3, cols);
  nmap_dst.create (rows * 3, cols);

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  tranformMapsKernel << < grid, block >> > (rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst, inverse);
  cudaSafeCall (hipGetLastError ());

  cudaSafeCall (hipDeviceSynchronize ());
}

void
pcl::device::tranformMaps (const MapArr& vmap_src, const MapArr& nmap_src, 
                           const Mat33& Rmat, const float3& tvec, 
                           MapArr& vmap_dst, MapArr& nmap_dst, const float3& newOrigin, const float3& objectCenter)
{
  int cols = vmap_src.cols ();
  int rows = vmap_src.rows () / 3;

  vmap_dst.create (rows * 3, cols);
  nmap_dst.create (rows * 3, cols);

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  tranformMapsKernel << < grid, block >> > (rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst, newOrigin, objectCenter);
  cudaSafeCall (hipGetLastError ());

  cudaSafeCall (hipDeviceSynchronize ());
}

namespace pcl
{
  namespace device
  {
    template<bool normalize>
    __global__ void
    resizeMapKernel (int drows, int dcols, int srows, const PtrStep<float> input, PtrStep<float> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= dcols || y >= drows)
        return;

      const float qnan = numeric_limits<float>::quiet_NaN ();

      int xs = x * 2;
      int ys = y * 2;

      float x00 = input.ptr (ys + 0)[xs + 0];
      float x01 = input.ptr (ys + 0)[xs + 1];
      float x10 = input.ptr (ys + 1)[xs + 0];
      float x11 = input.ptr (ys + 1)[xs + 1];

      if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
      {
        output.ptr (y)[x] = qnan;
        return;
      }
      else
      {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
          n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
      }
    }

    template<bool normalize>
    void
    resizeMap (const MapArr& input, MapArr& output)
    {
      int in_cols = input.cols ();
      int in_rows = input.rows () / 3;

      int out_cols = in_cols / 2;
      int out_rows = in_rows / 2;

      output.create (out_rows * 3, out_cols);

      dim3 block (32, 8);
      dim3 grid (divUp (out_cols, block.x), divUp (out_rows, block.y));
      resizeMapKernel<normalize><< < grid, block >> > (out_rows, out_cols, in_rows, input, output);
      cudaSafeCall ( hipGetLastError () );
      cudaSafeCall (hipDeviceSynchronize ());
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::resizeVMap (const MapArr& input, MapArr& output)
{
  resizeMap<false>(input, output);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::resizeNMap (const MapArr& input, MapArr& output)
{
  resizeMap<true>(input, output);
}

namespace pcl
{
  namespace device
  {

    template<typename T>
    __global__ void
    convertMapKernel (int rows, int cols, const PtrStep<float> map, PtrStep<T> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= cols || y >= rows)
        return;

      const float qnan = numeric_limits<float>::quiet_NaN ();

      T t;
      t.x = map.ptr (y)[x];
      if (!isnan (t.x))
      {
        t.y = map.ptr (y + rows)[x];
        t.z = map.ptr (y + 2 * rows)[x];
      }
      else
        t.y = t.z = qnan;

      output.ptr (y)[x] = t;
    }

	template<typename T>
    __global__ void
    convertErrorMapKernel (int rows, int cols, const PtrStep<float> map, PtrStep<T> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= cols || y >= rows)
        return;

      const float qnan = numeric_limits<float>::quiet_NaN ();

      T t;
      t.x = map.ptr (y)[x];
      if (!isnan (t.x))
      {
        t.y = map.ptr (y + rows)[x];
        t.z = map.ptr (y + 2 * rows)[x];
		t.w = map.ptr (y + 3 * rows)[x]; 
      }
      else
        t.y = t.z = t.w = qnan;

      output.ptr (y)[x] = t;
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<typename T> void
pcl::device::convert (const MapArr& vmap, DeviceArray2D<T>& output)
{
  int cols = vmap.cols ();
  int rows = vmap.rows () / 3;

  output.create (rows, cols);

  dim3 block (32, 8);
  dim3 grid (divUp (cols, block.x), divUp (rows, block.y));

  convertMapKernel<T><< < grid, block >> > (rows, cols, vmap, output);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

template<typename T> void
pcl::device::convertXYZI (const MapArr& errorMap, DeviceArray2D<T>& output)
{
  int cols = errorMap.cols ();
  int rows = errorMap.rows () / 4;

  output.create (rows, cols);

  dim3 block (32, 8);
  dim3 grid (divUp (cols, block.x), divUp (rows, block.y));

  convertErrorMapKernel<T><< < grid, block >> > (rows, cols, errorMap, output);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

template void pcl::device::convert (const MapArr& vmap, DeviceArray2D<float4>& output);
template void pcl::device::convert<float8>(const MapArr& vmap, DeviceArray2D<float8>& output);
template void pcl::device::convertXYZI (const MapArr& vmap, DeviceArray2D<float4>& output);

