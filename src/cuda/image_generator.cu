#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

using namespace pcl::device;

namespace pcl
{
  namespace device
  {
    struct ImageGenerator
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8
      };

      PtrStep<float> vmap;
      PtrStep<float> nmap;

      LightSource light;

      mutable PtrStepSz<uchar3> dst;

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= dst.cols || y >= dst.rows)
          return;

        float3 v, n;
        v.x = vmap.ptr (y)[x];
        n.x = nmap.ptr (y)[x];

        uchar3 color = make_uchar3 (0, 0, 0);

        if (!isnan (v.x) && !isnan (n.x))
        {
          v.y = vmap.ptr (y + dst.rows)[x];
          v.z = vmap.ptr (y + 2 * dst.rows)[x];

          n.y = nmap.ptr (y + dst.rows)[x];
          n.z = nmap.ptr (y + 2 * dst.rows)[x];

          float weight = 1.f;

          for (int i = 0; i < light.number; ++i)
          {
            float3 vec = normalized (light.pos[i] - v);

            weight *= fabs (dot (vec, n));
          }

          int br = (int)(205 * weight) + 50;
          br = max (0, min (255, br));
          color = make_uchar3 (br, br, br);
        }
        dst.ptr (y)[x] = color;
      }
    };

	struct ErrorImageGenerator
	{
	  enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8
      };

	  PtrStep<float> error;
	  mutable PtrStepSz<uchar3> dst;

	  __device__ __forceinline__ void
      operator () () const
      {
		int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= dst.cols || y >= dst.rows)
          return;

		uchar3 color = make_uchar3 (0, 0, 0);
		if(error.ptr(y)[x] == -1) {
			color = make_uchar3 (0, 0, 0);
		} else if(error.ptr(y)[x] < 6) {
			color = make_uchar3 (0, 0, 255);
		} else if(error.ptr(y)[x] < 12) {
			color = make_uchar3 (0, 128, 128);
		} else if(error.ptr(y)[x] < 18) {
			color = make_uchar3 (0, 255, 0);
		} else if(error.ptr(y)[x] < 24) {
			color = make_uchar3 (128, 128, 0);
		} else {
			color = make_uchar3 (255, 0, 0);
		}

		dst.ptr(y)[x] = color;
	  }

	};

	struct TSDFErrorImageGenerator
	{
	  enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8
      };

	  PtrStep<float> error;
	  mutable PtrStepSz<uchar3> dst;

	  __device__ __forceinline__ void
      operator () () const
      {
		int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= dst.cols || y >= dst.rows)
          return;

		uchar3 color = make_uchar3 (0, 0, 0);
		if(error.ptr(y)[x] == -1) {
			color = make_uchar3 (0, 0, 0);
		} else if(error.ptr(y)[x] < 0.4) {
			color = make_uchar3 (0, 0, 255);
		} else if(error.ptr(y)[x] < 0.8) {
			//Almost Grey
			color = make_uchar3 (0, 125, 128);
		} else if(error.ptr(y)[x] < 1.2) {
			color = make_uchar3 (0, 255, 0);
		} else if(error.ptr(y)[x] < 1.6) {
			color = make_uchar3 (128, 128, 0);
		} else {
			color = make_uchar3 (255, 0, 0);
		}

		dst.ptr(y)[x] = color;
	  }

	};

    __global__ void
    generateImageKernel (const ImageGenerator ig) {
      ig ();
    }

	__global__ void
	generateErrorImageKernel (const ErrorImageGenerator eig) {
	  eig ();
	}

	__global__ void
	generateTSDFErrorImageKernel (const TSDFErrorImageGenerator teig) {
	  teig ();
	}

  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::generateImage (const MapArr& vmap, const MapArr& nmap, const LightSource& light, 
                            PtrStepSz<uchar3> dst)
{
  ImageGenerator ig;
  ig.vmap = vmap;
  ig.nmap = nmap;
  ig.light = light;
  ig.dst = dst;

  dim3 block (ImageGenerator::CTA_SIZE_X, ImageGenerator::CTA_SIZE_Y);
  dim3 grid (divUp (dst.cols, block.x), divUp (dst.rows, block.y));

  generateImageKernel << < grid, block >> > (ig);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
}

void 
pcl::device::generateErrorImage(DeviceArray2D<float>& error, PtrStepSz<uchar3> dst) 
{
  ErrorImageGenerator eig;
  eig.error = error;
  eig.dst = dst;

  dim3 block (ErrorImageGenerator::CTA_SIZE_X, ErrorImageGenerator::CTA_SIZE_Y);
  dim3 grid (divUp (dst.cols, block.x), divUp (dst.rows, block.y));

  generateErrorImageKernel << < grid, block >> > (eig);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
}

void
pcl::device::generateTSDFErrorImage(DeviceArray2D<float>& error, PtrStepSz<uchar3> dst)
{
  TSDFErrorImageGenerator teig;
  teig.error = error;
  teig.dst = dst;

  dim3 block (ErrorImageGenerator::CTA_SIZE_X, ErrorImageGenerator::CTA_SIZE_Y);
  dim3 grid (divUp (dst.cols, block.x), divUp (dst.rows, block.y));

  generateTSDFErrorImageKernel << < grid, block >> > (teig);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
    __global__ void 
    paint3DViewKernel(const PtrStep<uchar3> colors, PtrStepSz<uchar3> dst, float colors_weight)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      
      if (x < dst.cols && y < dst.rows)
      {
        uchar3 value = dst.ptr(y)[x];        
        uchar3 color = colors.ptr(y)[x];

        if (value.x != 0 || value.y != 0 || value.z != 0)
        {
          float cx = value.x * (1.f - colors_weight) + color.x * colors_weight;
          float cy = value.y * (1.f - colors_weight) + color.y * colors_weight;
          float cz = value.z * (1.f - colors_weight) + color.z * colors_weight;

          value.x = min(255, max(0, __float2int_rn(cx)));
          value.y = min(255, max(0, __float2int_rn(cy)));
          value.z = min(255, max(0, __float2int_rn(cz)));
        }

        dst.ptr(y)[x] = value;
      }
    }
  }
}

void 
pcl::device::paint3DView(const PtrStep<uchar3>& colors, PtrStepSz<uchar3> dst, float colors_weight)
{
  dim3 block(32, 8);
  dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

  colors_weight = min(1.f, max(0.f, colors_weight));

  paint3DViewKernel<<<grid, block>>>(colors, dst, colors_weight);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());  
}